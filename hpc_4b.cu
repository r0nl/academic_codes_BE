
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 10

__global__ void matrixMult(int *a, int *b, int *c) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;
    for (int k = 0; k < N; k++) {
        sum += a[i * N + k] * b[k * N + j];
    }

    c[i * N + j] = sum;
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * N * sizeof(int);

    // Allocate memory for host matrices
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Initialize host matrices
    for (int i = 0; i < N * N; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // Allocate memory for device matrices
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy host matrices to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel with 2x2 threads per block
    clock_t start = clock();
    dim3 threadsPerBlock(2, 2);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
    matrixMult<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);
    clock_t end = clock();

    // Copy result from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print result
    printf("\nMatrix A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", a[i * N + j]);
        }
        printf("\n");
    }

    printf("\nMatrix B:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", b[i * N + j]);
        }
        printf("\n");
    }

    printf("\nResult Matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", c[i * N + j]);
        }
        printf("\n");
    }

    // Calculate elapsed time
    double elapsed_time = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("\n\nElapsed time: %f seconds\n", elapsed_time);

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
