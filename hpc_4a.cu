
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Number of elements in the vectors
#define N 1000000

// Kernel function to add vectors
__global__ void addVectors(float *a, float *b, float *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = N * sizeof(float);

    // Allocate memory for host vectors
    h_a = (float *)malloc(size);
    h_b = (float *)malloc(size);
    h_c = (float *)malloc(size);

    // Initialize host vectors
    srand(time(NULL)); // Seed the random number generator
    for (int i = 0; i < N; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    // Allocate memory for device vectors
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel with 1 block of 256 threads
    clock_t start = clock();
    addVectors<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    clock_t end = clock();

    // Copy result from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verify result
    int error = 0;
    for (int i = 0; i < N; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error: h_c[%d] = %f, h_a[%d] = %f, h_b[%d] = %f\n",
                   i, h_c[i], i, h_a[i], i, h_b[i]);
            error++;
            break;
        }
    }
    printf("Error: %d\n", error);

    // Calculate elapsed time
    double elapsed_time = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time: %f seconds\n", elapsed_time);

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
